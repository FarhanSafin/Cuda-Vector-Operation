#include "hip/hip_runtime.h"
﻿#include <stdio.h>

__global__ void dotProduct(float* A, float* B, float* C, float* D, float* result, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = (A[i] - B[i]) * (C[i] + D[i]);
    }
}

int main() {
    int n = 10000;
    float* h_A, * h_B, * h_C, * h_D, * h_result;
    float* d_A, * d_B, * d_C, * d_D, * d_result;
    size_t bytes = n * sizeof(float);

    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);
    h_D = (float*)malloc(bytes);
    h_result = (float*)malloc(bytes);

    for (int i = 0; i < n; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
        h_C[i] = i * 3;
        h_D[i] = i * 4;
    }

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);
    hipMalloc(&d_D, bytes);
    hipMalloc(&d_result, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_D, h_D, bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    dotProduct << <gridSize, blockSize >> > (d_A, d_B, d_C, d_D, d_result, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

    hipMemcpy(h_result, d_result, bytes, hipMemcpyDeviceToHost);

    hipEventRecord(start, 0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float totalTime;
    hipEventElapsedTime(&totalTime, start, stop);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    free(h_result);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_result);

    printf("Time took to run the actual kernel inside the GPU: %f ms\n", kernelTime);
    printf("Time it took to copy data in and out of the GPU: %f ms\n", kernelTime - totalTime);

    return 0;
}